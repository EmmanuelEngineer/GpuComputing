#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include "./include/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define TIMER_DEF struct timeval temp_1, temp_2
#define NAIVE_BLOCKSIZE 8

#define TIMER_START gettimeofday(&temp_1, (struct timezone *)0)

#define TIMER_STOP gettimeofday(&temp_2, (struct timezone *)0)

#define TIMER_ELAPSED ((temp_2.tv_sec - temp_1.tv_sec) + (temp_2.tv_usec - temp_1.tv_usec) / 1000000.0)

#define DBG_CHECK                                                  \
  if (verbose)                                                     \
  {                                                                \
    printf("DBG_CHECK: file %s at line %d\n", __FILE__, __LINE__); \
  }
#define DEBUG // without debug (with random imputs) the kernel does not work

#define NPROBS 5
#define STR(s) #s
#define XSTR(s) STR(s)
#define dtype float

#define PRINT_MATRIX(A, N, M, ST)         \
  {                                       \
    int i, j;                             \
    printf("%s:\n", (ST));                \
    for (i = 0; i < (N); i++)             \
    {                                     \
      printf("\t");                       \
      for (j = 0; j < (M); j++)           \
        printf("%6.3f ", A[i * (M) + j]); \
      printf("\n");                       \
    }                                     \
    printf("\n\n");                       \
  }

float matrix_error(int n, int m, const dtype *A, const dtype *B)
{
  int i, j;
  dtype error = (dtype)0;
  for (i = 0; i < n; i++)
    for (j = 0; j < m; j++)
      error += fabs(B[i * m + j] - A[i * m + j]);

  return (error);
}

#define BLOCK_ROWS 4
#define BLOCKSIZE 32
// #define BLOCKSIZE 1024
#define BLOCKEDGE(R) ((BLOCKSIZE) / (R)) // non usato
#define CEIL_DIV(N, D) (((N) % (D)) == 0) ? ((N) / (D)) : (((N) / (D)) + 1)

int verbose;

__global__ void naive_kernel(int N, int M, const dtype *A, dtype *B)
{
  // compute position in C that this thread is responsible for
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < N && y < M)
  {
    for (int i = 0; i < N; i++)
      for (int j = 0; j < M; j++)
        B[j * M + i] = A[i * M + j];
  }
}

__global__ void kernel_block_global(int N, int M, const dtype *A, dtype *B)
{
  // compute position in C that this thread is responsible for
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  int e_k, e_z;
  int block = blockDim.x;
  // `if` condition is necessary for when M or N aren't multiples of 32.
  if (x < N && y < M)
  {
    for (int i = 0; i < N; i++)
      for (int j = 0; j < M; j++)
      {
        e_k = (i + 1) + block;
        e_z = (j + 1) + block;
        for (int k = i; k < e_k && k < N; k++)
        {
          for (int z = j; z < e_z && z < M; z++)
            B[j * M + i] = A[i * M + j];
        }
      }
  }
}

__global__ void kernel_block_shared(int N, int M, const dtype *A, dtype *B)
{
  // compute position in B that this thread is responsible for
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  __shared__ dtype sharedA[BLOCKSIZE];
  __shared__ dtype sharedB[BLOCKSIZE];
  int matrixBlockStart = blockIdx.x * blockDim.x;
  int matrixBlockEnd = (blockIdx.x + 1) * blockDim.x;
  if (threadIdx.x == 0)
  {
    for (int i = matrixBlockStart; i < N; i++)
    {
      for (int j = 0; j < M; j++)
      {
        sharedA[threadIdx.x] = A[i * M + j];
      }
    }
  }
  __syncthreads();

  sharedA[threadIdx.x] += sharedB[threadIdx.x];

  for (int i = matrixBlockStart; i < N; i++)
  {
    for (int j = 0; j < M; j++)
    {
      B[j * M + i] = sharedA[threadIdx.x];
    }
  }
}

dtype *execute_kernel(int n, int m, dtype *A, void (*kernel)(int, int, const dtype *, dtype *), int blk_ratio, hipDeviceProp_t deviceProp, float *Bandwidth, float *CompTime, double *Flops)
{
  int grd_sizeX, grd_sizeY;
  int blk_sizeX, blk_sizeY;

  DBG_CHECK
  if (BLOCKSIZE % blk_ratio != 0)
  {
    fprintf(stderr, "Error: BLOCKSIZE (%d) is not divisible for blk_ratio (%d)\n", BLOCKSIZE, blk_ratio);
    exit(__LINE__);
  }
  DBG_CHECK
  // ---------------------------------
  int matrix_byte_dimension = deviceProp.sharedMemPerMultiprocessor / 2;
  int matrix_number_of_values = matrix_byte_dimension / sizeof(dtype);
  int number_of_blocks = matrix_number_of_values / BLOCKSIZE;
  char sw = (kernel == naive_kernel) ? '0' : ((kernel == kernel_block_global) ? '1' : ((kernel == kernel_block_shared) ? '2' : '3'));
  switch (sw)
  {
  case '0':
    blk_sizeX = BLOCKSIZE;
    grd_sizeX = CEIL_DIV(n, BLOCKSIZE);
    blk_sizeY = 1;
    grd_sizeY = 1;
    break;
  case '1':
    /* Modify here your kernel launch dimension */

    blk_sizeX = BLOCKSIZE;
    grd_sizeX = CEIL_DIV(n, BLOCKSIZE);
    blk_sizeY = 1;
    grd_sizeY = 1;
    break;
  case '2':
    /* Modify here your kernel launch dimension */
    blk_sizeX = BLOCKSIZE;
    grd_sizeX = CEIL_DIV(n, BLOCKSIZE);
    blk_sizeY = 1;
    grd_sizeY = 1;
    break;
  case '3':
    /* Modify here your kernel launch dimension */
    blk_sizeX = BLOCKSIZE;
    grd_sizeX = CEIL_DIV(n, BLOCKSIZE);
    blk_sizeY = 1;
    grd_sizeY = 1;
    break;
  }
  // ---------------------------------

  DBG_CHECK
  // ------------------- allocating GPU vectors ----------------------
  dtype *dev_A, *dev_B;
  checkCudaErrors(hipMalloc(&dev_A, n * m * sizeof(dtype)));
  checkCudaErrors(hipMalloc(&dev_B, n * m * sizeof(dtype)));
  size_t bandwidth_numerator = (n * m) * sizeof(dtype);

  DBG_CHECK
  // ----------------- copy date from host to device -----------------
  checkCudaErrors(hipMemcpy(dev_A, A, n * m * sizeof(dtype), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(dev_B, 0, n * m * sizeof(dtype)));

  DBG_CHECK
  // ---------- compute GPU_tmp_b with the reduction kernel ----------
  TIMER_DEF;
  TIMER_START;

  {
    dim3 block_size(blk_sizeX, blk_sizeY, 1);
    dim3 grid_size(grd_sizeX, grd_sizeY, 1);
    printf("%d: block_size = (%d, %d), grid_size = (%d, %d)\n", __LINE__, block_size.x, block_size.y, grid_size.x, grid_size.y);
    kernel<<<grid_size, block_size>>>(n, m, (const dtype *)dev_A, dev_B);
  }

  checkCudaErrors(hipDeviceSynchronize());
  TIMER_STOP;
  *CompTime += TIMER_ELAPSED;
  *Bandwidth = bandwidth_numerator / ((*CompTime) * 1e+9);
  *Flops = (n * m) / ((*CompTime) * 1e+9);

  DBG_CHECK
  // --------------- copy results from device to host ----------------

  dtype *GPU_B = (dtype *)malloc(sizeof(dtype) * n * m);
  checkCudaErrors(hipMemcpy(GPU_B, dev_B, n * m * sizeof(dtype), hipMemcpyDeviceToHost));

  if (verbose > 0)
    PRINT_MATRIX(GPU_B, n, m, "GPU_B form execute_kernel")

  DBG_CHECK
  checkCudaErrors(hipFree(dev_A));
  checkCudaErrors(hipFree(dev_B));
  DBG_CHECK
  return (GPU_B);
}

void usage(char *bin_name, int exit_faulier)
{
  fprintf(stderr, "Usage: %s -n <n> -m <m> [-c] [-v]\n", bin_name);
  fprintf(stderr, "Where mandatory inputs are:\n");
  fprintf(stderr, "\t-n\trepresents rows number of the two matrices\n");
  fprintf(stderr, "\t-m\trepresents columns number of the two matrices\n\n");

  fprintf(stderr, "And optional inputs are:\n");
  fprintf(stderr, "\t-c\tif provided it enables CPU compare computation\n");
  fprintf(stderr, "\t-v\tif provided it enables verbose prints\n\n");
  exit(exit_faulier);
}

#define CHECKRTYPE(exitval, opt)               \
  {                                            \
    if (exitval == gread)                      \
      prexit("Unexpected option -%c!\n", opt); \
    else                                       \
      gread = !exitval;                        \
  }

int main(int argc, char *argv[])
{

  printf("====================================== Problem computations ======================================\n");
  // =========================================== Set-up the problem ============================================

  int p = 0;
  int n, m;
  char input;
  int cpuOn_flag = 0;
  while ((input = getopt(argc, argv, "n:m:cv")) != EOF)
  {

    switch (input)
    {
    // BC approx  c param is the costanst used in Bader stopping cretierion
    case 'n':
      sscanf(optarg, "%d", &n);
      if (n <= 0)
      {
        fprintf(stderr, "Error: n value must be a positive integer (%d provided)\n", n);
        usage(argv[0], __LINE__);
      }
      else
      {
        p |= 1;
      }
      break;
    case 'm':
      sscanf(optarg, "%d", &m);
      if (m <= 0)
      {
        fprintf(stderr, "Error: m value must be a positive integer (%d provided)\n", n);
        usage(argv[0], __LINE__);
      }
      else
      {
        p |= 2;
      }
      break;
    case 'c':
      cpuOn_flag = 1;
      break;
    case 'v':
      verbose = 1;
      break;
    case 'h':
      usage(argv[0], __LINE__);
    case '?':
      fprintf(stderr, "Error: unrecognized parameter (%c)\n\n", input);
      usage(argv[0], __LINE__);
    }
#undef CHECKRTYPE
  }

  if (p != 3)
  {
    fprintf(stderr, "Error: -n and -m parameters are mandatory\n\n");
    usage(argv[0], __LINE__);
  }

  // ---------------- set-up the problem size -------------------

  //   printf("e = %d --> n = k = m = 2^(e/2) = %d\n", e, n);
  //   printf("alpha = %f, beta = %f\n", alpha, beta);
  printf("CPU_ON = %d\n", cpuOn_flag);
  printf("verbose = %d\n", verbose);
  printf("dtype = %s\n", XSTR(dtype));

  // ======================================== Get the device properties ========================================
  printf("======================================= Device properties ========================================\n");

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  int dev;
  hipDeviceProp_t deviceProp;
  for (dev = 0; dev < deviceCount; ++dev)
  {
    hipSetDevice(dev);
    hipGetDeviceProperties(&deviceProp, dev);

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);

    printf("  Memory Bus Width:                              %d bit\n",
           deviceProp.memoryBusWidth);

    printf("  Peak Memory Bandwidth:                     %7.3f GB/s\n",
           2.0 * deviceProp.memoryClockRate * (deviceProp.memoryBusWidth / 8) / 1.0e6);

    printf("  (%03d) Multiprocessors, (%03d) CUDA Cores/MP:    %d CUDA Cores\n",
           deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
               deviceProp.multiProcessorCount);

    printf("  Peak Arithmetic Intensity:                     %7.3f GFLOPS/s\n",
           2.0 * deviceProp.memoryClockRate * (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount) / 1.0e6);
  }

  // ------------------ set-up the timers ---------------------

  TIMER_DEF;
  const char *lables[NPROBS] = {"CPU check", "Naive Kernel", "kernel_block_global", "kernel_block_shared", "kernel_1"};
  float errors[NPROBS], Times[NPROBS], Bandwidths[NPROBS], error;
  double Flops[NPROBS];
  for (int i = 0; i < NPROBS; i++)
  {
    Bandwidths[i] = 0.0;
    errors[i] = -1.0;
    Flops[i] = 0.0;
    Times[i] = 0.0;
  }

  DBG_CHECK
  // ------------------- set-up the problem -------------------

  dtype *A, *GPU_B, *CPU_B;
  A = (dtype *)malloc(sizeof(dtype) * n * m);
  CPU_B = (dtype *)malloc(sizeof(dtype) * n * m);
  GPU_B = (dtype *)malloc(sizeof(dtype) * n * m);

  time_t t;
  srand((unsigned)time(&t));

  for (int i = 0; i < (n * m); i++)
  {
    A[i] = ((dtype)(i / m) / (dtype)m) + 1.0f;
  }

#ifdef DEBUG
  if (verbose > 0)
  {
    PRINT_MATRIX(A, n, m, "A")
  }
#endif
  // ======================================== Running the computations =========================================

  /* [ ... ]
   */

  DBG_CHECK
  // ========================== CPU computation =========================
  if (cpuOn_flag)
  {

    TIMER_START;
    for (int i = 0; i < n; i++)
      for (int j = 0; j < m; j++)
        CPU_B[j * m + i] = A[i * m + j];
    TIMER_STOP;

    Times[0] = TIMER_ELAPSED;
    errors[0] = 0.0f;
    Bandwidths[0] = -1.0f;
    Flops[0] = (n * m) / (Times[0] * 1e+9);

    if (verbose > 0)
      PRINT_MATRIX(CPU_B, n, m, "CPU_C")

    printf("CPU time: %lf\n", Times[0]);
  }
  else
  {
    Times[0] = -1.0f;
    errors[0] = -1.0f;
    Bandwidths[0] = -1.0f;
    Flops[0] = -1.0f;
  }

  DBG_CHECK
  // =========================== GPU naive Kernel ===========================
  printf("=========================== GPU naive Kernel ===========================\n");

  DBG_CHECK
  GPU_B = execute_kernel(n, m, A, naive_kernel, 1, deviceProp, &Bandwidths[1], &Times[1], &Flops[1]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_B, GPU_B)) : (error = 0.0f);
  errors[1] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_B, n, m, "GPU_B")
  printf("Error equal to %lf\n", error);

  free(GPU_B);

  // =========================== GPU Kernel 1 ===========================
  printf("=========================== GPU Kernel 1 ===========================\n");

  GPU_B = execute_kernel(n, m, A, kernel_block_global, 1, deviceProp, &Bandwidths[2], &Times[2], &Flops[2]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_B, GPU_B)) : (error = 0.0f);
  errors[2] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_B, n, m, "GPU_B")

  free(GPU_B);

  // =========================== GPU Kernel 2 ===========================
  printf("=========================== GPU Kernel 2 ===========================\n");

  GPU_B = execute_kernel(n, m, A, kernel_block_shared, 1, deviceProp, &Bandwidths[3], &Times[3], &Flops[3]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_B, GPU_B)) : (error = 0.0f);
  errors[3] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_B, n, m, "GPU_B")

  free(GPU_B);

  // =========================== GPU Kernel 3 ===========================
  printf("=========================== GPU Kernel 3 ===========================\n");

  GPU_B = execute_kernel(n, m, A, kernel_block_shared, 1, deviceProp, &Bandwidths[4], &Times[4], &Flops[4]);

  // ------------- Compare GPU and CPU solution --------------

  (cpuOn_flag) ? (error = matrix_error(n, m, CPU_B, GPU_B)) : (error = 0.0f);
  errors[4] = error;

  if (verbose > 0)
    PRINT_MATRIX(GPU_B, n, m, "GPU_B")

  free(GPU_B);

  printf("\n\n");
  if (!(cpuOn_flag))
    printf("CPU check not lunched!!\n");
  printf("Solution\n %9s\t%9s\t%9s\t%16s\t%16s\n", "type", "error", "time (s)", "flops (GFLOPS/s)", "bandwidth (GB/s)");
  for (int i = 0; i < NPROBS; i++)
  {
    if ((i != 6))
      printf("%12s:\t%9.6f\t%9.6f\t%16.6lf\t%16.6f\n", lables[i], errors[i], Times[i], Flops[i], Bandwidths[i]);
  }
  printf("\n");

  printf("GPU times: n*m Kernel1_time Kernel1_flops Kernel2_time Kernel2_flops ... on stderr\n");
  fprintf(stderr, "%d, ", n * m);
  for (int i = 1; i < NPROBS; i++)
    fprintf(stderr, "%f, %f, ", Times[i], Flops[i]);
  fprintf(stderr, "\n");

  return (0);
}
/*  */