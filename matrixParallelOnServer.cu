
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#define ThreadsperBlock 64 //We have a maximum of 64 cuda cores/real threads in a block
#define BlocksToUse 1 // the system has a maximum of 56 Streaming multiprocessor/real blocks

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++)
	y[i] = x[i] + y[i];
}


int main(int argc, char **argv)
{
	int N;
	N = atoi(argv[1]);
	N = N<<20;
	printf("N:%d\n",N);
	float *x, *y;
	hipEvent_t start,stop;
// Allocate Unified Memory accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));
// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = rand()/100*0.5f;
		y[i] = rand()/100*0.3f;
	}
// Separate threads per block each threds has to compute N/(tpb*b)
    int arrayBlockSize = N/ BlocksToUse*ThreadsperBlock;
    


// Run kernel on 1M elements on the GPU
	hipEventCreate(&start);
	add<<<BlocksToUse,ThreadsperBlock>>>(N, x, y);
	hipEventCreate(&stop);

// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
// Check for errors (all values should be 3.0f)
	float milliseconds;
	hipEventElapsedTime(&milliseconds,start,stop);
	printf("Kernel Time: %f ms\n",milliseconds);
	float maxError = 0.0f;
	for (int  i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	std::cout << "Max error: " << maxError << std::endl;
// Free memory
	hipFree(x);
	hipFree(y);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
